#include "hip/hip_runtime.h"
using namespace std;
#include<cstdio>
#include <cstdlib>
#include <string>
#include <iostream>
#include <vector>
#include <fstream>
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <unordered_map>
#include <climits>
#include <chrono>
#include <iomanip>
// 301 191 161  92
#define blockSize 256
#define TRY 1
#define TARGET 161
#define origin 103
#define TH_PER_BLOCK 1000
#define VERTICES 50000//2000  
/*

CUDA Avg Time (ms): 3883.098632813
103 -> 137 -> 165 -> 191 -> 161
 distance: 794
*/         //number of vertices
#define DENSITY 16              //minimum number of edges per vertex. DO NOT SET TO >= VERTICES
#define MAX_WEIGHT 1000000      //max edge length + 1
#define INF_DIST 1000000000     //"infinity" initial value of each node
#define CPU_IMP 1               //number of Dijkstra implementations (non-GPU)
#define GPU_IMP 1               //number of Dijkstra implementations (GPU)
#define THREADS 2               //number of OMP threads
#define RAND_SEED 1234          //random seed
#define THREADS_BLOCK 512
using namespace std;
void sortPart(int *array,int *array2,int start,int n)
{
    for(int i=0;i<n;i++)
    {
        int t=array[start+i];
        int t2=array2[start+i];

        for(int j=i;j<n;j++)
        {
           if(t>array[start+j]) 
           {
            array[start+i]=array[start+j];
            array2[start+i]=array2[start+j];
            array[start+j]=t;
            array2[start+j]=t2;
            t=array[start+i];
            t2=array2[start+i];
           }
        }
    }
}
int getLenFromFile()
{
    int numStations=VERTICES;
    int i=0;
     std::ifstream fileConnections("NewYork/NewYork_Edgelist.csv");
    if (!fileConnections.is_open()) {
        std::cerr << "Error opening file: " << "Data/connections.txt" << std::endl;
    }

    std::string t1;
    getline(fileConnections,t1);
    int k=0;
    int t=0;
    for( i=0; true; i++){

        int source, target,c2;
        char c;
        float a,b,weight;

        fileConnections>>a>>c>>b>>c>>source>>c>>target>>c>>c2>>c>>weight;
        getline(fileConnections,t1);

        if(source<numStations&&target<numStations)
        {  
            
        }
        else break;
        // std::cout<<source<<" "<<target<<" "<<weight<<std::endl;
    }
    fileConnections.close();
    return i;
}
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void printPath(const int * parent, int dest) {
    if (parent[dest] == -1) {
        cout << dest;
        return;
    }
    printPath(parent, parent[dest]);
    cout << " -> " << dest;
}
void setIntArrayValue(int* in_array, int array_size, int init_value) {
    int i;
    for (i = 0; i < array_size; i++) {
        in_array[i] = init_value;
    }
}

/*  Initialize elements of a 1D data_t array with an initial value   */
void setDataArrayValue(int* in_array, int array_size, int init_value) {
    int i;
    for (i = 0; i < array_size; i++) {
        in_array[i] = init_value;
    }
}

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, ": %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__global__ void reduce6NoVisitedV2(int *g_idata,int *distance,int *g_out,  const int n) {
    extern __shared__ int sdata;
    sdata=1;
    extern __shared__ int indexes1[VERTICES/TH_PER_BLOCK+2];
    int tip=blockIdx.x * blockDim.x + threadIdx.x;
    if(tip*sdata+sdata<n&&tip<n)
    {
        indexes1[threadIdx.x]= tip;
        //  __syncthreads();
           while (sdata<n) { 
                int i= 2*sdata*(threadIdx.x);
                if(i+sdata<n&&
                  distance[g_idata[indexes1[i]]]>distance[g_idata[indexes1[i+sdata]]])
                     indexes1[i]=indexes1[i+sdata];
                if(threadIdx.x==0)
                 sdata*=2;
                __syncthreads();
            }
    }
    if(threadIdx.x==0)
    {
        g_out[blockIdx.x]=g_idata[indexes1[0]];
    }
}
__global__ void closestNodeCUDA(int* node_dist, int* visited_node, int* global_closest, int num_vertices) {
    int dist = INF_DIST + 1;
    int node = -1;
    int i;
    for (i = 0; i < num_vertices; i++) {
        if ((node_dist[i] < dist) && (visited_node[i] != 1)) {
            dist = node_dist[i];
            node = i;
        }
    }
    global_closest[0] = node;
    visited_node[node] = 1;
}

void minItera(int *M, int*N1,int *MIN,int *W,int *E,int *NW)
{
    int N=VERTICES;
    int t=origin;
    cout<<" t="<<t<<" M="<<M[t]<<" n1="<<N1[t]<<endl;
    if (t < N -10&& M[t] == 1&&N1[t]!=-1) {
        cout<<" usao"<<endl;
        int tempT=t;
        for(int kh=t+1;kh<N-10;kh++)
        {
            if(N1[kh]!=-1)
            {
                tempT=kh;
                break;
            }
        }
        cout<<" n1="<<N1[t]<<" n12="<<N1[tempT]<<endl;
        for (int z = N1[t]; z < N1[tempT]; z++) { 
                int oldNW = NW[E[z]] ;
                cout<<" z="<<z<<" ez="<<E[z]<<" nwez="<<NW[E[z]]<<" w="<<W[z]<<" t"<<t<<" newt="<<NW[t]<<endl;
               if (oldNW > NW[t] + W[ z]) {
                    NW[E[z]]= NW[t] + W[ z];
                }

                if (*MIN> NW[E[z]]) {
                    *MIN=NW[E[z]];
                }
                cout<<" z="<<z<<endl;
                // N1[t] = z;
                break;
            
        }
    }
}

__global__ void minimumKernel(int *M, int *N1, int *NW, int *E, int *W, int *MIN, int N) {
    //NW node distance
    //M visited node
    //W graph
    //n1 is a copy of n
    //n array that has index from which index in E does start the edges
    int t = threadIdx.x + blockIdx.x * blockDim.x;

    if (t < N -1&& M[t] == 1&&N1[t]!=-1) {
        int tempT=t;
        for(int kh=t+1;kh<N-1;kh++)
        {
            if(N1[kh]!=-1)
            {
                tempT=kh;
                break;
            }
        }
        for (int z = N1[t]; z < N1[tempT]; z++) { 
            if(M[E[z]]!=1)
            {
                int tempnw=NW[t]+W[z];
                if(NW[t]==INT_MAX)
                    tempnw=INT_MAX;
                int oldNW = atomicMin(&NW[E[z]], tempnw);
                if (oldNW > tempnw) {
                    atomicExch(&NW[E[z]],tempnw);
                }

                int minVal = atomicMin(MIN, NW[E[z]]);
                if (minVal > NW[E[z]]) {
                    atomicExch(MIN, NW[E[z]]);
                }

                N1[t] = z;
                break;
            }
        }
    }
}

/*
Algorithm 9: SET_FLAG (M, NW, MIN)
BEGIN
t=getThreadID
if(M[t]!=1 &&NW[t]==MIN) then
M[t]=1
if end
END
*/
__global__ void setFlag(int *m,int *nw,int *min)
{
    int t = threadIdx.x + blockIdx.x * blockDim.x;
    if(t<VERTICES)
    {
         if(m[t]!=1&&nw[t]==*min)
            m[t]=1;
        __syncthreads();
        if(t==0)
        *min=INT_MAX;
    }
   
}

__global__ void cudaRelax(int* graph, int* node_dist, int* parent_node, int* visited_node, int* global_closest) {
     int next = blockIdx.x*blockDim.x + threadIdx.x;    //global ID
     if(true||next<VERTICES)
     {
        int source = global_closest[0];

        int edge = graph[source*VERTICES + next];
        int new_dist = node_dist[source] + edge;

        if ((edge != 0) &&
            (visited_node[next] != 1) &&
            (new_dist < node_dist[next])) {
            node_dist[next] = new_dist;
            parent_node[next] = source;
        }
     }
}

__global__ void reduce6(int *g_idata, int *g_odata,int *visired,unsigned int n) {
    extern __shared__ int sdata;
    sdata=1;
    extern __shared__ int indexes[TH_PER_BLOCK];
    int tip=blockIdx.x * blockDim.x + threadIdx.x;
    if(tip*sdata+sdata<VERTICES&&tip<VERTICES)
    {
        //zbog ovog koristimo 2x vise threads, potential memory size improvements
        indexes[threadIdx.x]= tip;
        //  __syncthreads();
           while (sdata<TH_PER_BLOCK) { 
                int i= 2*sdata*(threadIdx.x);
                if(i+sdata<TH_PER_BLOCK&&
                  visired[indexes[i+sdata]]!=1&&
                  g_idata[indexes[i]]>g_idata[indexes[i+sdata]])
                     indexes[i]=indexes[i+sdata];
                else if(i+sdata<TH_PER_BLOCK&&visired[indexes[i+sdata]]!=1&&visired[indexes[i]]==1)
                     indexes[i]=indexes[i+sdata];
                   
                if(threadIdx.x==0)
                 sdata*=2;
                __syncthreads();
            }
    }
    if(threadIdx.x==0)
    {
        g_odata[blockIdx.x ]=indexes[0];
        //ovo ispod treba prebaciti izvan kernela jer min tek dobijem kad prodjem kroz sve blokove
        // visired[indexes[0]]=1;
    }
}
__global__ void reduce6NoVisited(int *g_idata,int *distance,int *visited,unsigned int n) {
    extern __shared__ int sdata;
    sdata=1;
    extern __shared__ int indexes2[TH_PER_BLOCK];
    int tip=blockIdx.x * blockDim.x + threadIdx.x;
    if(tip*sdata+sdata<n&&tip<n)
    {
        indexes2[threadIdx.x]= tip;
        //  __syncthreads();
           while (sdata<n) { 
                int i= 2*sdata*(threadIdx.x);
                if(i+sdata<n&&
                  distance[g_idata[indexes2[i]]]>distance[g_idata[indexes2[i+sdata]]])
                     indexes2[i]=indexes2[i+sdata];
                if(threadIdx.x==0)
                 sdata*=2;
                __syncthreads();
            }
    }
    if(threadIdx.x==0)
    {
        g_idata[0]=g_idata[indexes2[0]];
        visited[g_idata[0]]=1;
    }
}

int * createConnectionMatrix() {

    std::unordered_map<std::string, int> stationIndices; 
    std::vector<std::string> stations; 
    std::vector<std::vector<int>> adjacencyMatrix; 

    std::ifstream fileStations("Data/stations.txt");
    if (!fileStations.is_open()) {
        std::cerr << "Error opening file: " << "Data/stations.txt" << std::endl;
    }

    for(int i = 0; i<376; i++){
       int nodeId;
        std::string nodeName;
        fileStations >> nodeId;
        std::getline(fileStations >> std::ws, nodeName);
        stations.push_back(nodeName);
    }
    fileStations.close();


    int numStations = stations.size();
    adjacencyMatrix.resize(numStations, std::vector<int>(numStations, 0));

    std::ifstream fileConnections("Data/connections.txt");
    if (!fileConnections.is_open()) {
        std::cerr << "Error opening file: " << "Data/connections.txt" << std::endl;
    }

    for(int i=0; i<=933; i++){
        int source, target, weight;
        fileConnections >> source >> target >> weight;
        adjacencyMatrix[source][target] = weight;
    }
    fileConnections.close();
    int *t=(int*)calloc(adjacencyMatrix.size()*adjacencyMatrix[0].size(),sizeof( int* ));
    for(int i=0;i<adjacencyMatrix.size();i++)
    {
        for(int j=0;j<adjacencyMatrix[i].size();j++)
        t[i*adjacencyMatrix.size()+j]=adjacencyMatrix[i][j];
    }
    printf("neki=%zd %zd\n",adjacencyMatrix.size(),adjacencyMatrix[0].size());
    return  t;
}

int* createConnectionMatrix2() {

    int numStations = VERTICES;
    int * adjacencyMatrix=(int*)malloc(sizeof(int)*numStations*numStations);
    std::ifstream fileConnections("NewYork/NewYork_Edgelist.csv");
    if (!fileConnections.is_open()) {
        std::cerr << "Error opening file: " << "Data/connections.txt" << std::endl;
    }

    std::string t1;
    getline(fileConnections,t1);
    for(int i=0; true; i++){

        int source, target,c2;
        char c;
        float a,b,weight;

        fileConnections>>a>>c>>b>>c>>source>>c>>target>>c>>c2>>c>>weight;
        getline(fileConnections,t1);

        if(source<numStations&&target<numStations)
     {   adjacencyMatrix[source*numStations+target] = weight;
        adjacencyMatrix[target*numStations+source] = weight;
    }
        else break;
        // std::cout<<source<<" "<<target<<" "<<weight<<std::endl;
    }
    fileConnections.close();

    return  adjacencyMatrix; 
}

int* createConnectionMatrix3(int *e,int *w,int *n1, int*n) {

    int numStations = VERTICES;
    int len=getLenFromFile();
    for(int i=0;i<numStations;i++)
    {
        n[i]=-1;
        n1[i]=-1;
    }
    cout<<" len="<<len<<endl;
    
    std::ifstream fileConnections("NewYork/NewYork_Edgelist.csv");
    if (!fileConnections.is_open()) {
        std::cerr << "Error opening file: " << "Data/connections.txt" << std::endl;
    }

    std::string t1;
    getline(fileConnections,t1);
    int t=0,old;
    for(int i=0; true; i++){

        int source, target,c2;
        char c;
        float a,b,weight;

        fileConnections>>a>>c>>b>>c>>source>>c>>target>>c>>c2>>c>>weight;
        getline(fileConnections,t1);
        if(i==0)
        {
            old=source;
            t=0;
        }

        if(source<numStations&&i<VERTICES)
        {  
            if(target>numStations-1)
            {
                i--;
                continue;
            }
            if(old!=source)
            {
                sortPart(w,e,t,i-t);
                n[old]=t;
                n1[old]=t;
                t=i;
                old=source;
                //14 14 926 77 637 926 77 135 234 135 234 344 802 344 802 1252 61 61 77 564 proslo 
            }
            e[i]=target;
            w[i]=weight;
        }
        else{
            n[old]=t;
            n1[old]=t;
            t=i;
            n[old+1]=t;
            n1[old+1]=t;
            cout<<" kraje var"<<old<<" t="<<t<<" tart="<<target<<" sou="<<source<<endl;
            break;
        } 
        // std::cout<<source<<" "<<target<<" "<<weight<<std::endl;
    }
    fileConnections.close();
    return  n; 
}

using namespace std;
int blockNum(int vertecies,int threads)
{
    int block=1;
    int tempBlock=VERTICES-TH_PER_BLOCK;
    while(tempBlock>0)
    {
        block++;
        tempBlock-=TH_PER_BLOCK;
    }
    return block;
}


int  main(){
    cout<<" dgao";
//     for(int i=0;i<VERTICES;i++)
// {
// cout<<graf[1*VERTICES+i]<<" ";
// }
    //performance measure, time
    float elapsed_exec;  
    hipEvent_t exec_start, exec_stop; 
    hipEventCreate(&exec_start);
    hipEventCreate(&exec_stop);
 //declare variables and allocate memory
 cout<<"dsfadfa";
 long long int tempSize=VERTICES;
    long long int graph_size = tempSize*tempSize*sizeof(int);             //memory in B required by adjacency matrix representation of graph
     long long int int_array       = VERTICES*sizeof(int);                         //memory in B required by array of vertex IDs. Vertices have int IDs.
     long long int data_array      = VERTICES*sizeof(int);                      //memory in B required by array of vertex distances (depends on type of data used)
    // int* graph       = (int*)malloc(graph_size);                  //graph itself
    int* node_dist  = (int*)malloc(VERTICES*sizeof(int));        
    cout<<" alocatae1 11 ";          //distances from source indexed by node ID
    int* parent_node   = (int*)malloc(VERTICES*sizeof(int));                       //number of edges per node indexed by node ID
    int* visited_node= (int*)malloc(VERTICES*sizeof(int));                       //number of edges per node indexed by node ID
    int* visited_node2= (int*)malloc(VERTICES*sizeof(int));                       //number of edges per node indexed by node ID
    int* visited_node1= (int*)malloc(VERTICES*sizeof(int));                      //pseudo-bool if node has been visited indexed by node ID
    // int *pn_matrix      = (int*)malloc((CPU_IMP+GPU_IMP)*int_array);    //matrix of parent_node arrays (one per each implementation)
    // int* dist_matrix = (int*)malloc((CPU_IMP + GPU_IMP)*data_array);

    int * nn1=(int*)malloc(sizeof(int)*VERTICES);
     createConnectionMatrix3(visited_node,node_dist,parent_node,nn1);
    printf("Variables created, allocated\n");

    //CUDA mallocs
    int* e;
    int* w;
    int* n;
    int* n1;
    int* minOut;
    int *reduction,*reduction1;
    hipMalloc((void**)&e, graph_size);
    hipMalloc((void**)&w, data_array);
    hipMalloc((void**)&n, int_array);
    hipMalloc((void**)&n1, int_array);
    hipMalloc((void**)&reduction1, blockNum(VERTICES,TH_PER_BLOCK)*sizeof(int));
    hipMalloc((void**)&reduction, blockNum(VERTICES,TH_PER_BLOCK)*sizeof(int));
    hipMalloc((void**)&minOut, int_array);
    int block=1;
    // for(int i=0;i<20;i++)
    // {
    //     cout<<nn1[i]<<" ";
    // }
    // cout<<endl<<";;;;;;;;;;;;;;;;;;;;;"<<endl;
    // for(int i=0;i<20;i++)
    // {
    //     cout<<node_dist[i]<<" ";
    // }
    int tempBlock=VERTICES-TH_PER_BLOCK;
    while(tempBlock>0)
    {
        block++;
        tempBlock-=TH_PER_BLOCK;
    }
    int* closest_vertex = (int*)malloc(sizeof(int)*block);
    int* gpu_closest_vertex,*gpu_m,*gpu_nw;
    closest_vertex[0] = INT_MAX;
    float totalTime=0.0;
    cout<<"proslo "<<endl;
    //    node_dist[origin]=0;
         cout<<" dga---"<<endl;
        hipMalloc((void**)&gpu_closest_vertex, (sizeof(int)*block));
        hipMalloc((void**)&gpu_m, int_array);
        hipMalloc((void**)&gpu_nw, int_array);
       CUDA_SAFE_CALL(hipMemcpy(gpu_closest_vertex, closest_vertex, sizeof(int), hipMemcpyHostToDevice));
        (hipMemcpy(e, visited_node, int_array, hipMemcpyHostToDevice));
        setIntArrayValue(visited_node1, VERTICES, 0); 
        visited_node1[origin]=1;
        cout<<" dos =="<<endl;
        (hipMemcpy(gpu_m, visited_node1, int_array, hipMemcpyHostToDevice));
        hipMemcpy(w, node_dist, data_array, hipMemcpyHostToDevice);
        hipMemcpy(n,nn1, int_array, hipMemcpyHostToDevice);
        hipMemcpy(n1, parent_node, int_array, hipMemcpyHostToDevice);
        setDataArrayValue(visited_node2,VERTICES,INT_MAX);
        visited_node2[origin]=0;
        hipMemcpy(gpu_nw,visited_node2, int_array, hipMemcpyHostToDevice);
        // cout<<"koperia"<<endl;
        // for(int j=0;j<VERTICES;j++)
        // {
        //     if(j==103)
        //     cout<<"("<<setw(3)<<j<<","<<setw(3)<<parent_node[j]<<") ";
        //     else
        //     cout<<nn1[j]<<" ";
        // }
        // minItera(visited_node1,nn1,closest_vertex,node_dist,visited_node,visited_node2);
        cout<<" moe in="<<*closest_vertex<<endl;
        dim3 gridMin(1, 1, 1);
        dim3 blockMin(1, 1, 1);
        dim3 gridRelax(VERTICES / THREADS_BLOCK, 1, 1);
        dim3 blockRelax(THREADS_BLOCK, 1, 1);   
    for(int k=0;k<TRY;k++)
    {        
        printf("Krece exec\n");
        hipEventRecord(exec_start);
        for (int i = 0; true; i++)
        {
            // cout<<" end"<<endl<<endl;
            //__global__ void minimumKernel(int *M, int *N1, int *NW, int *E, int *W, int *MIN, int N) {
                // (hipMemcpy(closest_vertex, gpu_closest_vertex, sizeof(int), hipMemcpyDeviceToHost));
                // cout<<" closest node="<<*closest_vertex<<endl;
                 (hipMemcpy(visited_node, gpu_nw, sizeof(int)*VERTICES, hipMemcpyDeviceToHost));
            //  if(i==2)
            //  for(int u=0;u<VERTICES;u++){
            //     if(visited_node[u]!=2147483647)
            //     // cout<<"x ";
            //     // else
            //     cout<<visited_node[u]<<" ";
            //  }
            minimumKernel<<<block,TH_PER_BLOCK>>>(gpu_m,n1,gpu_nw,e,w,gpu_closest_vertex,VERTICES);
             (hipMemcpy(closest_vertex, gpu_closest_vertex, sizeof(int), hipMemcpyDeviceToHost));
                // cout<<" closest node1="<<*closest_vertex<<endl;
                if(*closest_vertex==INT_MAX)
                break;
            setFlag<<<block,TH_PER_BLOCK>>>(gpu_m,gpu_nw,gpu_closest_vertex);
            
            
        }
        hipEventRecord(exec_stop);
        hipEventSynchronize(exec_stop);
        hipEventElapsedTime(&elapsed_exec, exec_start, exec_stop);        //elapsed execution time
        printf("\n\nCUDA Time (ms): %7.9f\n", elapsed_exec);
        totalTime+=elapsed_exec;
    }
    printf("\n\nCUDA Avg Time (ms): %7.9f\n", totalTime/TRY);
    (hipMemcpy(node_dist, gpu_nw, data_array, hipMemcpyDeviceToHost));
    (hipMemcpy(parent_node, n, int_array, hipMemcpyDeviceToHost));
    (hipMemcpy(visited_node, n1, int_array, hipMemcpyDeviceToHost));
    // for (int i = 0; i < VERTICES; i++) {  
    //     // cout<<node_dist[i]<<" ";              //record resulting parent array and node distance
    //     pn_matrix[VERTICES + i] = parent_node[i];
    //     dist_matrix[VERTICES + i] = node_dist[i];
    // }
    // printPath(parent_node,TARGET);
    printf("\n distance: %d %d",node_dist[TARGET],TARGET);//794
       //free memory
    (hipFree(e));
    (hipFree(w));
    (hipFree(n));
    (hipFree(n1));
    (hipFree(gpu_closest_vertex));
    (hipFree(reduction1));
    (hipFree(reduction));
    // free(graf);
    free(closest_vertex);
    free(node_dist);
    free(parent_node);
    free(visited_node);
    // free(pn_matrix);
    // free(dist_matrix);
    return 0;
}
